
#include <hip/hip_runtime.h>
#include <stdio.h>

//for matrices of A(2x3) and  B(3x2)//
#define SIZE 2
#define SIZE1 3  

__global__ void matMult(int * matProd, int * matA, int * matB)
{
        int row = blockIdx.x;
        int col = threadIdx.x;

        int tmpSum = 0;;
        if (row < SIZE && col < SIZE)
        {
                for (int i=0; i<SIZE1; ++i)
                        tmpSum += matA[row*SIZE1 + i] * matB[i*SIZE + col];
                matProd[row*SIZE + col] = tmpSum;
        }
}

int main()
{
        // initialize, aalocate and define host memory
        int matA[SIZE*SIZE1] = { 0 };
        int matB[SIZE1*SIZE] = { 0 };
        int matProd[SIZE*SIZE] = { 0 };

        //set Matrix A
        for (int i=0; i<SIZE; i++)
        {
                for (int j=0; j<SIZE1; j++)
                {
                        matA[i*SIZE1 + j] = i+j;    
                }
        }

        //set Matrix B
        for (int i=0; i<SIZE1; i++)
        {
                for (int j=0; j<SIZE; j++)
                {  
                        matB[i*SIZE + j] = i-j;
                }
        }


        // initialize and allocate device memory
        int * dev_matProd, * dev_matA, * dev_matB;
        hipMalloc((void **)&dev_matA, SIZE*SIZE1*sizeof(int));
        hipMalloc((void **)&dev_matB, SIZE1*SIZE*sizeof(int));
        hipMalloc((void **)&dev_matProd, SIZE*SIZE*sizeof(int));


        // copy data to device memory
        hipMemcpy((void *)dev_matA, (void *)matA, SIZE*SIZE1*sizeof(int),
                        hipMemcpyHostToDevice);
        hipMemcpy((void *)dev_matB, (void *)matB, SIZE1*SIZE*sizeof(int),
                        hipMemcpyHostToDevice);

        matMult<<<SIZE,SIZE>>>(dev_matProd, dev_matA, dev_matB);
        
        // check for successful thread execution
        if (hipDeviceSynchronize() != hipSuccess)
        {
                printf("Error\n");
                return -1;
        }

        // copy results from device to host memory
        hipMemcpy(matProd, dev_matProd, SIZE*SIZE*sizeof(int),
                        hipMemcpyDeviceToHost);

        
        printf("Matrix A \n");
        for(int i =0;i<SIZE;i++)
        {
            for(int j=0;j<SIZE1;j++)
            {
                printf("%d \t",matA[i*SIZE1+j]);
            }
            printf("\n");
        }

        printf("Matrix B \n");
        for(int i =0;i<SIZE1;i++)
        {
            for(int j=0;j<SIZE;j++)
            {
                printf("%d \t",matB[i*SIZE+j]);
            }
            printf("\n");
        }


        printf("Matrix Product \n");
        for(int i =0;i<SIZE;i++)
        {
            for(int j=0;j<SIZE;j++)
            {
                printf("%d \t",matProd[i*SIZE+j]);
            }
            printf("\n");
        }





        return 0;
}